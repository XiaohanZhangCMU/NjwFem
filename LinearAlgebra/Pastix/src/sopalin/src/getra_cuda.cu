#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "redefine_functions.h"

#ifdef TYPE_COMPLEX
#  include <hip/hip_complex.h>
#  ifdef PREC_DOUBLE
#    define FLOAT hipDoubleComplex
#    define MULT(a, b)  hipCmul(a,b)
#    define ADD(a, b)   hipCadd(a,b)
#  else
#    define FLOAT hipFloatComplex
#    define MULT(a,b)  hipCmulf(a,b)
#    define ADD(a, b)  hipCaddf(a,b)
#  endif
#else /* not TYPE_COMPLEX */
#  define MULT(a,b) ((a) * (b))
#  define ADD(a, b) ((a) + (b))
#  ifdef PREC_DOUBLE
#    define FLOAT double
#  else
#    define FLOAT float
#  endif
#endif

#define TILE_DIM   16

#include "getra_cuda.h"

__global__ void getra_cuda_kernel(FLOAT *A, int lda, FLOAT * B, int ldb, int N)
{
  __shared__ FLOAT block[TILE_DIM][TILE_DIM+1];
	
  // read the matrix tile into shared memory
  unsigned int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  unsigned int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  if((xIndex < N) && (yIndex < N))
    {
      unsigned int index_in = yIndex * lda + xIndex;
      block[threadIdx.y][threadIdx.x] = A[index_in];
    }

  __syncthreads();

  // write the transposed matrix tile to global memory
  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  if((xIndex < N) && (yIndex < N))
    {
      unsigned int index_out = yIndex * ldb + xIndex;
      B[index_out] = block[threadIdx.x][threadIdx.y];
    }
}



extern "C" void
getra_cuda(FLOAT *A, int lda, FLOAT * B, int ldb, int N)
{
  dim3 threads( TILE_DIM, TILE_DIM );
  dim3 grid (N/threads.x + (N%threads.x != 0), 1+N/threads.y + (N%threads.y != 0));

  getra_cuda_kernel<<< grid, threads >>>(A, lda, B, ldb, N);
}
