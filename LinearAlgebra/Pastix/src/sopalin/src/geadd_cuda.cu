#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "redefine_functions.h"

#ifdef TYPE_COMPLEX
#  include <hip/hip_complex.h>
#  ifdef PREC_DOUBLE
#    define FLOAT hipDoubleComplex
#    define MULT(a, b)  hipCmul(a,b)
#    define ADD(a, b)   hipCadd(a,b)
#  else
#    define FLOAT hipFloatComplex
#    define MULT(a,b)  hipCmulf(a,b)
#    define ADD(a, b)  hipCaddf(a,b)
#  endif
#else /* not TYPE_COMPLEX */
#  define MULT(a,b) ((a) * (b))
#  define ADD(a, b) ((a) + (b))
#  ifdef PREC_DOUBLE
#    define FLOAT double
#  else
#    define FLOAT float
#  endif
#endif

#include "geadd_cuda.h"

#define geadd_cuda_kernel_nn PASTIX_PREFIX_F(geadd_cuda_kernel_nn)
#define geadd_cuda_kernel_tn PASTIX_PREFIX_F(geadd_cuda_kernel_tn)
#define geadd_cuda_kernel_nt PASTIX_PREFIX_F(geadd_cuda_kernel_nt)
#define geadd_cuda_kernel_tt PASTIX_PREFIX_F(geadd_cuda_kernel_tt)

__global__ void geadd_cuda_kernel_nn(int m, int n,
				     FLOAT alpha, FLOAT *a, int lda,
				     FLOAT beta,  FLOAT *b, int ldb)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  int index_a =i+j*lda;
  int index_b =i+j*ldb;
  if ( i < m && j < n )
    b[index_b]= ADD( MULT(alpha,a[index_a]), MULT(beta,b[index_b]));
}

__global__ void geadd_cuda_kernel_nt(int m, int n,
				     FLOAT alpha, FLOAT *a, int lda,
				     FLOAT beta,  FLOAT *b, int ldb)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  int index_a =i+j*lda;
  int index_b =j+i*ldb;
  if ( i < m && j < n)
    b[index_b]= ADD( MULT(alpha,a[index_a]), MULT(beta,b[index_b]));
}



__global__ void geadd_cuda_kernel_tn(int m, int n,
				     FLOAT alpha, FLOAT *a, int lda,
				     FLOAT beta,  FLOAT *b, int ldb)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  int index_a =j+i*lda;
  int index_b =i+j*ldb;
  if ( i < m && j < n)
    b[index_b]= ADD( MULT(alpha,a[index_a]), MULT(beta,b[index_b]));
}

__global__ void geadd_cuda_kernel_tt(int m, int n,
				     FLOAT alpha, FLOAT *a, int lda,
				     FLOAT beta,  FLOAT *b, int ldb)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  int index_a =j+i*lda;
  int index_b =j+i*ldb;
  if ( i < m && j < n)
    b[index_b]= ADD( MULT(alpha,a[index_a]), MULT(beta,b[index_b]));
}
extern "C" void
geadd_cuda(char * transa, char * transb,
	   int m, int n,
	   FLOAT alpha, FLOAT *a, int lda,
	   FLOAT beta,  FLOAT *b, int ldb)
{
  dim3 threads( 16, 4 );
  dim3 grid (m/threads.x + (m%threads.x != 0), n/threads.y + (n%threads.y != 0));

  if (*transa == 'N') {
    if (*transb == 'N') {
      geadd_cuda_kernel_nn<<< grid, threads >>>(m, n,
						alpha, a, lda,
						beta,  b, ldb);
    }
    else {
      geadd_cuda_kernel_nt<<< grid, threads >>>(m, n,
						alpha, a, lda,
						beta,  b, ldb);
    }
  }
  else {
    if (*transb == 'N') {
      geadd_cuda_kernel_tn<<< grid, threads >>>(m, n,
						alpha, a, lda,
						beta,  b, ldb);
    }
    else {
      geadd_cuda_kernel_tt<<< grid, threads >>>(m, n,
						alpha, a, lda,
						beta,  b, ldb);
    }  }
}
